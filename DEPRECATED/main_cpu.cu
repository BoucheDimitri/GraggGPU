#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <math.h>
/**************************************************************
The code in time.h is a part of a course on cuda taught by its authors:
Lokman A. Abbas-Turki
**************************************************************/
#include "timer.h"


//Function to print a small square matrix of floats on host
void print_matrix(float *c, int n) {

	for (int i=0; i<n; i++){

    		for(int j=0; j<n; j++) {

         		printf("%f     ", c[n * i + j]);
        	}

    		printf("\n");
 	}	
}

//Function to print a small vector of floats on host
void print_vector(float *c, int m, int n) {

	for (int i=0; i<m; i++){

		printf("%f     ", c[i]);

    		printf("\n");
 	}	
}



// Fill c with arrow matrix generated from vectors a and b
// Not very useful actually for our problem
void generate_arrow(float *a, float *b, float *c, float gamma, int n) {
	
	int j = 0; 

	// Fill the arrow
	for (int i=0; i<n; i ++){
		
		if (i<n-1) {

		// Iterate over the last column of c
		c[n - 1 + i*n] = b[i];
		
		// Iterate over the last row of c
		c[n * (n-1) + i] = b[i];

		// Iterate over the diagonal of c
		c[n*i + j] = a[i];
		j ++; 

		}
	}

	// Fill last element of diagonal with gamma
	c[(n-1) * (n-1)] = gamma;
}


void vector_square(float *b, float *bsqr, int n){

	for (int i=0; i<n-1; i++){
		bsqr[i] = b[i] * b[i]; 
	}
}



// Compute f (the spectral function) at a given point x
float spectral_func(float *a, 
		    float *bsqr, 
		    float x, 
		    float gamma, 
	    	    int n) {
	
	float sum = 0;

	for (int i=0; i<n-1; i++){
		sum += bsqr[i] / (a[i] - x);
	}
	
	return x - gamma + sum;
}


// Compute f' (the prime derivative of the spectral function) at a given point x
float spectral_func_prime(float *a, 
			  float *bsqr, 
			  float x, 
			  int n) {
	
	float sum = 0;

	for (int i=0; i<n-1; i++){

		int ai_local = a[i];
		sum += bsqr[i] / ((ai_local - x) * (ai_local - x));
	}
	
	return 1 + sum;
}


// Compute the interior versions of sigma
float interior_sigma(float *a, 
		     float *bsqr, 
		     float x, 
		     float gamma, 
		     int k,
		     int n) {
	
	float sum = 0;

	float ak_local = a[k];
	float ak_minus1_local = a[k-1]; 

	for (int i=0; i<n-1; i++) {
		
		//Use the registers
		float ai_local = a[i];
		
		float num = bsqr[i] * (ai_local - ak_minus1_local) * (ai_local - ak_local);
		
		float deno = (ai_local - x) * (ai_local - x) 
		        * (ai_local - x);
		
		sum +=  num / deno;
	}

	float term1 = 3 * x - gamma - ak_local - ak_minus1_local;

	return term1 + sum;
}


// Compute the interior version of alpha
float interior_alpha(float sigma, float x, float ak, float ak_minus1){

	return sigma / ((ak_minus1 - x) * (x - ak));
}



// Compute the interior version of beta
float interior_beta(float fprime, float f, float x, float ak, float ak_minus1){

	float fac = (1 / (ak_minus1 - x) + 1 / (ak - x)); 
	return fprime - fac * f; 
	
}


// Computation of the update (delta) for the interior intervals
float interior_delta(float f, float alpha, float beta){

	float term1 = 2 * f / beta;
	float term2 = 2 * alpha / beta;
	float deno = 1 + sqrtf(1 + term1 * term2);
	return term1 / deno; 
}


//function to find the zero within the interval (a[k], a[k-1])
float interior_zero_finder(float *a, 
			   float *bsqr, 
			   float gamma, 
			   float x, 
			   int k, 
			   int n, 
			   int maxit, 
			   float epsilon){

	int i = 0;
	// To guarantee entry in the loop
	float f = 2 * sqrtf(epsilon); 
	while ((i < maxit) && (f*f > epsilon)){
		// Computation of sigma(x), solution of system (5) in page 7 (12 in the pdf) of the article
		float sig = interior_sigma(a, bsqr, x, gamma, k, n);
		float ak_local = a[k]; 
		float ak_minus1_local = a[k - 1]; 
		// Computation of alpha(x), see definition (7) of the article in page 8 (13 in the pdf)
		float alpha = interior_alpha(sig, x, ak_local, ak_minus1_local);
		// Computation of spectral_func(x)
		f = spectral_func(a, bsqr, x, gamma, n);
		// Computation of spectral_func_prime(x)
		float fprime = spectral_func_prime(a, bsqr, x, n);
		// Computation of beta(x), see definition (8) of the article in page 8 (13 in the pdf)
		float beta = interior_beta(fprime, f, x, ak_local, ak_minus1_local);
		// Computation of delta(x), see definition (9) of the article in page 8 (13 in the pdf)
		float delta = interior_delta(f, alpha, beta);
		// Update of x
		x -= delta;
		i ++; 
	}
	return x; 
}

int main (void) {


	// Declare vectors
	float *a, *b, *bsqr, *x0_vec, *xstar_vec, *c; 


	// Gamma
	float gamma = 1; 


	// Size of arrow matrix
	int n = 100;


	//Maximum number of iterations
	int maxit = 10000; 


	//Stopping criterion
	float epsilon = 0.000001;  
	

	// Memory allocation
	a = (float*)malloc((n-1)*sizeof(float));
	b = (float*)malloc((n-1)*sizeof(float));
	bsqr = (float*)malloc((n-1)*sizeof(float));
	c = (float*)malloc(n*n*sizeof(float));
	x0_vec = (float*)malloc(n*sizeof(float));
	xstar_vec = (float*)malloc(n*sizeof(float));

	
	// Create instance of class Timer
	Timer Tim;
	

	// Fill the vectors a and b (arbitrarily for now)
	for (int i=0; i<n; i++){
		a[i] = 2 * n - i;
	}

	for (int i=0; i<n-1; i++){
		b[i] = n - i;
	}


	// We take the middle of the intervals as initial value 
	//(as advised in the paper at the beginning of  page 8 (13 of the pdf) 
	for (int i=1; i<n-1; i++){
		x0_vec[i] = (a[i-1] + a[i]) / 2 ;
	}
	
	//Arbitrary filling of the edges values (TO REPLACE BY INITIAL VALUES FROM THE PAPER)
	x0_vec[0] = a[0] + 5;
	x0_vec[n-1] = a[n-2] - 5; 


	// Fill c with arrow matrix generated from a and b
	//generate_arrow(a, b, c, gamma, n);

	// Print c (not very necessary actually)
	//printf("The arrow matrix : \n");
	//print_matrix(c, n);

	
	// Start timer
	Tim.start();


	//Compute square of b
	vector_square(b, bsqr, n);

	
	// Find all n-2 interior roots
	for (int k=1; k<n-1; k++) {

		xstar_vec[k] = interior_zero_finder(a, bsqr, gamma, x0_vec[k], k, n, maxit, epsilon);
	}


	// End timer
	Tim.add();
	

	// Print the first zeros
	// Number of roots to display
	int m = 10;
	printf("\n");
	printf("The first %i greater resulting roots (eigen values) are : \n", m);
	print_vector(xstar_vec, m, n);
	
	// Print how long it took
	printf("CPU timer for root finding (CPU-GPU and GPU-CPU transfers included) : %f s\n",
		(float)Tim.getsum());


	// Free memory on CPU
	free(a);
	free(b);
	free(bsqr);
	free(c);
	free(x0_vec); 
	free(xstar_vec);
	
}

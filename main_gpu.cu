#include "hip/hip_runtime.h"
#include <stdio.h>

/**************************************************************
The code in time.h is a part of a course on cuda taught by its authors:
Lokman A. Abbas-Turki
**************************************************************/
#include "timer.h"


//Function to print a small square matrix of floats on host
void print_matrix(float *c, int n) {

	for (int i=0; i<n; i++){

    		for(int j=0; j<n; j++) {

         		printf("%f     ", c[n * i + j]);
        	}

    		printf("\n");
 	}	
}

//Function to print a small vector of floats on host
void print_vector(float *c, int m, int n) {

	for (int i=0; i<m; i++){

		printf("%f     ", c[i]);

    		printf("\n");
 	}	
}



// Fill c with arrow matrix generated from vectors a and b
// Not very useful actually for our problem
void generate_arrow(float *a, float *b, float *c, float gamma, int n) {
	
	int j = 0; 

	// Fill the arrow
	for (int i=0; i<n; i ++){
		
		if (i<n-1) {

		// Iterate over the last column of c
		c[n - 1 + i*n] = b[i];
		
		// Iterate over the last row of c
		c[n * (n-1) + i] = b[i];

		// Iterate over the diagonal of c
		c[n*i + j] = a[i];
		j ++; 

		}
	}

	// Fill last element of diagonal with gamma
	c[(n-1) * (n-1)] = gamma;
}



// Kernel for computing the square of a vector
// We actually only need the square of b in the computations 
// Thus it is better to compute it once and for all
__global__ void square_kernel(float *bGPU, float *bsqrGPU, int n){
	
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	
	while(idx < n){
	bsqrGPU[idx] = bGPU[idx] * bGPU[idx];
	idx += gridDim.x * blockDim.x;
	}
}



// Device function for computing f (the spectral function) at a given point x
__device__ float spectral_func(float *aGPU, 
			       float *bsqrGPU, 
			       float x, 
			       float gamma, 
			       int n) {
	
	float sum = 0;

	for (int i=0; i<n-1; i++){
		sum += bsqrGPU[i] / (aGPU[i] - x);
	}
	
	return x - gamma + sum;
}


// Device function for computing f' (the prime derivative of the spectral function) at a given point x
__device__ float spectral_func_prime(float *aGPU, 
			       	     float *bsqrGPU, 
			             float x, 
			             int n) {
	
	float sum = 0;

	for (int i=0; i<n-1; i++){

		int ai_local = aGPU[i];
		sum += bsqrGPU[i] / ((ai_local - x) * (ai_local - x));
	}
	
	return 1 + sum;
}


// Device function to compute the interior versions of sigma
__device__ float interior_sigma(float *aGPU, 
		       		float *bsqrGPU, 
		                float x, 
		                float gamma, 
				int k,
		                int n) {
	
	float sum = 0;

	//Use the registers
	float ak_local = aGPU[k];
	float ak_minus1_local = aGPU[k-1]; 

	for (int i=0; i<n-1; i++) {
		
		//Use the registers
		float ai_local = aGPU[i];
		
		float num = bsqrGPU[i] * (ai_local - ak_minus1_local) * (ai_local - ak_local);
		
		float deno = (ai_local - x) * (ai_local - x) 
		        * (ai_local - x);
		
		sum +=  num / deno;
	}

	float term1 = 3 * x - gamma - ak_local - ak_minus1_local;

	return term1 + sum;
}


// Interior version for computation of alpha
__device__ float interior_alpha(float sigma, float x, float ak, float ak_minus1){

	return sigma / ((ak_minus1 - x) * (x - ak));
}



// Interior version for computation of beta
__device__ float interior_beta(float fprime, float f, float x, float ak, float ak_minus1){

	float fac = (1 / (ak_minus1 - x) + 1 / (ak - x)); 
	return fprime - fac * f; 
	
}


// Computation of the update (delta) on device
__device__ float interior_delta(float f, float alpha, float beta){

	float term1 = 2 * f / beta;
	float term2 = 2 * alpha / beta;
	float deno = 1 + sqrtf(1 + term1 * term2);
	return term1 / deno; 
}


// device function to find the zero within the interval (a[k], a[k-1])
__device__ float interior_zero_finder(float *aGPU, 
			   	      float *bsqrGPU, 
			              float gamma, 
			   	      float x, 
			   	      int k, 
			   	      int n, 
			   	      int maxit, 
			   	      float epsilon){

	int i = 0;
	// To guarantee entry in the loop
	float f = 2 * sqrtf(epsilon); 
	while ((i < maxit) && (f*f > epsilon)){
		// Computation of sigma(x), solution of system (5) in page 7 (12 in the pdf) of the article
		float sig = interior_sigma(aGPU, bsqrGPU, x, gamma, k, n);
		float ak_local = aGPU[k]; 
		float ak_minus1_local = aGPU[k - 1]; 
		// Computation of alpha(x), see definition (7) of the article in page 8 (13 in the pdf)
		float alpha = interior_alpha(sig, x, ak_local, ak_minus1_local);
		// Computation of spectral_func(x)
		f = spectral_func(aGPU, bsqrGPU, x, gamma, n);
		// Computation of spectral_func_prime(x)
		float fprime = spectral_func_prime(aGPU, bsqrGPU, x, n);
		// Computation of beta(x), see definition (8) of the article in page 8 (13 in the pdf)
		float beta = interior_beta(fprime, f, x, ak_local, ak_minus1_local);
		// Computation of delta(x), see definition (9) of the article in page 8 (13 in the pdf)
		float delta = interior_delta(f, alpha, beta);
		// Update of x
		x -= delta;
		i ++; 
	}
	return x; 
}
	

// Kernal to find the zeros (only the interior ones for now)   
__global__ void find_zeros_kernel(float *aGPU, 
				  float *bsqrGPU, 
				  float *xstart_vecGPU, 
				  float *xvecGPU, 
				  float gamma, 
				  int n, 
				  int maxit, 
				  float epsilon) {

	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	// IMPORTANT : n-2 and not n to consider only the interior intervals
	// TO BE MODIFIED 
	while(idx < n-2){
		
		// Initial value
		float x = xvecGPU[idx + 1]; 
		// Each core gets an interior interval and finds the unique zero within
		xstart_vecGPU[idx + 1] = interior_zero_finder(aGPU, bsqrGPU, gamma, x, idx + 1, n, maxit, epsilon); 
		// In case n - 2 > gridDim.x * blockDim.x
		idx += gridDim.x * blockDim.x;
	}
}


// KERNEL FOR TESTING, TO BE REMOVED, IGNORE
__global__ void test_all_kernel(float *aGPU, 
				float *bsqrGPU, 
				float *yvecGPU, 
				float *xvecGPU, 
				float gamma, 
				int n) {


	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	while(idx < n-2){
		float x_local = xvecGPU[idx + 1]; 
		float sig = interior_sigma(aGPU, bsqrGPU, x_local, gamma, idx + 1, n);
		float ak_local = aGPU[idx + 1]; 
		float ak_minus1_local = aGPU[idx]; 
		float alpha = interior_alpha(sig, x_local, ak_local, ak_minus1_local);
		float f = spectral_func(aGPU, bsqrGPU, x_local, gamma, n);
		float fprime = spectral_func_prime(aGPU, bsqrGPU, x_local, n);
		float beta = interior_beta(fprime, f, x_local, ak_local, ak_minus1_local);
		float delta = interior_delta(f, alpha, beta);
		yvecGPU[idx + 1] = delta;
		idx += gridDim.x * blockDim.x;
	}
}


int main (void) {


	// Declare vectors
	float *a, *b, *bsqr, *x0_vec, *xstar_vec, *c; 


	// Gamma
	float gamma = 1; 


	// Size of arrow matrix
	int n = 10;


	//Maximum number of iterations
	int maxit = 10000; 


	//Stopping criterion
	float epsilon = 0.000001;  
	

	// Memory allocation
	a = (float*)malloc((n-1)*sizeof(float));
	b = (float*)malloc((n-1)*sizeof(float));
	bsqr = (float*)malloc((n-1)*sizeof(float));
	c = (float*)malloc(n*n*sizeof(float));
	x0_vec = (float*)malloc(n*sizeof(float));
	xstar_vec = (float*)malloc(n*sizeof(float));

	
	// Create instance of class Timer
	Timer Tim;
	

	// Fill the vectors a and b (arbitrarily for now)
	for (int i=0; i<n; i++){
		a[i] = 2 * n - i;
	}

	for (int i=0; i<n-1; i++){
		b[i] = n - i;
	}


	// We take the middle of the intervals as initial value 
	//(as advised in the paper at the beginning of  page 8 (13 of the pdf) 
	for (int i=1; i<n-1; i++){
		x0_vec[i] = (a[i-1] + a[i]) / 2 ;
	}
	
	//Arbitrary filling of the edges values (TO REPLACE BY INITIAL VALUES FROM THE PAPER)
	x0_vec[0] = a[0] + 5;
	x0_vec[n-1] = a[n-2] - 5; 


	// Fill c with arrow matrix generated from a and b
	//generate_arrow(a, b, c, gamma, n);

	// Print c (not very necessary actually)
	//printf("The arrow matrix : \n");
	//print_matrix(c, n);

	
	// Declare vectors on GPU
	float *aGPU, *bGPU, *bsqrGPU, *x0_vecGPU, *xstar_vecGPU;

	// Create memory space for vectors on GPU
	hipMalloc(&aGPU, (n-1)*sizeof(float));
	hipMalloc(&bGPU, (n-1)*sizeof(float));
	hipMalloc(&bsqrGPU, (n-1)*sizeof(float));
	// The initial values
	hipMalloc(&x0_vecGPU, n*sizeof(float));
	// Container for the results
	hipMalloc(&xstar_vecGPU, n*sizeof(float));
	

	// Start timer
	// We time also the transfer time
	Tim.start();


	// Transfers on GPU
	hipMemcpy(aGPU, a, (n-1)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(bGPU, b, (n-1)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(x0_vecGPU, x0_vec, n*sizeof(float), hipMemcpyHostToDevice);


	//Compute square of b on GPU
	square_kernel <<<1024, 512>>> (bGPU, bsqrGPU, n);


	// Find interior zeros on GPU
	find_zeros_kernel<<<1024, 512>>> (aGPU, 
					  bsqrGPU, 
					  xstar_vecGPU, 
					  x0_vecGPU, 
					  gamma, 
					  n,
					  maxit, 
					  epsilon); 


	// Transfer results on CPU to print it
	hipMemcpy(xstar_vec, xstar_vecGPU, n*sizeof(float), hipMemcpyDeviceToHost);


	// End timer
	Tim.add();
	

	// Print the first zeros
	// Number of roots to display
	int m = 10;
	printf("\n");
	printf("The first %i greater resulting roots (eigen values) are : \n", m);
	print_vector(xstar_vec, m, n);

	
	// Print how long it took
	printf("CPU timer for root finding (CPU-GPU and GPU-CPU transfers included) : %f s\n",
		(float)Tim.getsum());



	// Free memory on GPU
	hipFree(aGPU);
	hipFree(bGPU);
	hipFree(bsqrGPU);
	hipFree(x0_vecGPU); 
	hipFree(xstar_vecGPU); 


	// Free memory on CPU
	free(a);
	free(b);
	free(bsqr);
	free(c);
	free(x0_vec); 
	free(xstar_vec);
	
}


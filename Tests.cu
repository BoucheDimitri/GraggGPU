
#include <hip/hip_runtime.h>
#include <stdio.h>


//Function to print a small square matrix of floats
void print_matrix(float *c, int n) {

	for (int i=0; i<n; i++){

    		for(int j=0; j<n; j++) {

         		printf("%f     ", c[n * i + j]);
        	}

    		printf("\n");
 	}	
}

//Function to print a small vector of floats
void print_vector(float *c, int n) {

	for (int i=0; i<n; i++){

		printf("%f     ", c[i]);

    		printf("\n");
 	}	
}



// Fill c with arrow matrix generated from vectors a and b
void generate_arrow(float *a, float *b, float *c, float gamma, int n) {
	
	int j = 0; 

	// Fill the arrow
	for (int i=0; i<n; i ++){
		
		if (i<n-1) {

		// Iterate over the last column of c
		c[n - 1 + i*n] = b[i];
		
		// Iterate over the last row of c
		c[n * (n-1) + i] = b[i];

		// Iterate over the diagonal of c
		c[n*i + j] = a[i];
		j ++; 

		}
	}

	// Fill last element of diagonal with gamma
	c[(n-1) * (n-1)] = gamma;
}



// Kernel for computing the square of a vector
// The square of b is needed during several computations 
// for all subproblems, so better to compute it once and for all
__global__ void square_kernel(float *bGPU, float *bsqrGPU, int n){
	
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	
	while(idx < n){
	bsqrGPU[idx] = bGPU[idx] * bGPU[idx];
	idx += gridDim.x * blockDim.x;
	}
}











int main (void) {

	// Declare vectors
	float *a, *b, *bsqr, *c;

	// Size of arrow matrix
	int n = 10;
	
	// Declare and reference gamma
	float gamma = 1;
	
	// Memory allocation
	a = (float*)malloc((n-1)*sizeof(float));
	b = (float*)malloc((n-1)*sizeof(float));
	bsqr = (float*)malloc((n-1)*sizeof(float));
	c = (float*)malloc(n*n*sizeof(float));

	// Fill the vectors
	for (int i=0; i<n; i++){
		a[i] = 20 - i;
	}

	for (int i=0; i<n-1; i++){
		b[i] = 10 - i;
	}

	// Fill c with arrow matrix generated from a and b
	generate_arrow(a, b, c, gamma, n);

	// Print c
	//print_matrix(c, n);

	
	// Declare vectors on GPU
	float *aGPU, *bGPU, *bsqrGPU;

	// Create memory space for vectors on GPU
	hipMalloc(&aGPU, (n-1)*sizeof(float));
	hipMalloc(&bGPU, (n-1)*sizeof(float));
	hipMalloc(&bsqrGPU, (n-1)*sizeof(float));

	// Transfer on GPU
	hipMemcpy(aGPU, a, (n-1)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(bGPU, b, (n-1)*sizeof(float), hipMemcpyHostToDevice);

	//Test square kernel
	square_kernel <<<1024, 512>>> (bGPU, bsqrGPU, n);

	hipMemcpy(bsqr, bsqrGPU, (n-1)*sizeof(float), hipMemcpyDeviceToHost);
	
	print_vector(bsqr, n-1);
	


	// Free memory on GPU
	hipFree(aGPU);
	hipFree(bGPU);
	hipFree(bsqrGPU);


	// Free memory on CPU
	free(a);
	free(b);
	free(bsqr);
	free(c);
	
}


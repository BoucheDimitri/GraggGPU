
#include <hip/hip_runtime.h>
#include <stdio.h>


//Function to print a small square matrix of floats
void print_matrix(float *c, int n) {

	for (int i=0; i<n; i++){

    		for(int j=0; j<n; j++) {

         		printf("%f     ", c[n * i + j]);
        	}

    		printf("\n");
 	}	
}

//Function to print a small vector of floats
void print_vector(float *c, int n) {

	for (int i=0; i<n; i++){

		printf("%f     ", c[i]);

    		printf("\n");
 	}	
}



// Fill c with arrow matrix generated from vectors a and b
void generate_arrow(float *a, float *b, float *c, float gamma, int n) {
	
	int j = 0; 

	// Fill the arrow
	for (int i=0; i<n; i ++){
		
		if (i<n-1) {

		// Iterate over the last column of c
		c[n - 1 + i*n] = b[i];
		
		// Iterate over the last row of c
		c[n * (n-1) + i] = b[i];

		// Iterate over the diagonal of c
		c[n*i + j] = a[i];
		j ++; 

		}
	}

	// Fill last element of diagonal with gamma
	c[(n-1) * (n-1)] = gamma;
}



// Kernel for computing the square of a vector
// The square of b is needed during several computations 
// for all subproblems, so better to compute it once and for all at the beginning
__global__ void square_kernel(float *bGPU, float *bsqrGPU, int n){
	
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	
	while(idx < n){
	bsqrGPU[idx] = bGPU[idx] * bGPU[idx];
	idx += gridDim.x * blockDim.x;
	}
}



// Device function for computing f(*xGPU)
__device__ float spectral_func(float *aGPU, 
			       float *bsqrGPU, 
			       float *xGPU, 
			       float *gammaGPU, 
			       int n) {
	
	float sum = 0;
	
	//For the use of registers
	float xGPU_local = *xGPU;

	for (int i=0; i<n-1; i++){
		sum += bsqrGPU[i] / (aGPU[i] - xGPU_local);
	}
	
	return xGPU_local - *gammaGPU + sum;
}


// Device function for computing f'(*xGPU)
__device__ float spectral_func_prime(float *aGPU, 
			       	     float *bsqrGPU, 
			             float *xGPU, 
			             int n) {
	
	float sum = 0;
	
	//For the use of registers
	float xGPU_local = *xGPU;

	for (int i=0; i<n-1; i++){

		int ai_local = aGPU[i];
		sum += bsqrGPU[i] / ((ai_local - xGPU_local) * (ai_local - xGPU_local));
	}
	
	return 1 + sum;
}





// Kernel associated with spectral_func device function
__global__ void spectral_func_kernel(float *aGPU, 
				     float *bsqrGPU, 
				     float *yvecGPU, 
				     float *xvecGPU, 
				     float *gammaGPU, 
				     int n) {

	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	while(idx < n){

	yvecGPU[idx] = spectral_func(aGPU, bsqrGPU, &(xvecGPU[idx]), gammaGPU, n);
	idx += gridDim.x * blockDim.x;

	}
}



// Device function to compute the interior versions of sigma
__device__ float sigma_interior(float *aGPU, 
		       		float *bsqrGPU, 
		                float *xGPU, 
		                float *gammaGPU, 
				int k,
		                int n) {
	
	float sum = 0;

	//Use the registers
	float xGPU_local = *xGPU;
	float ak_local = aGPU[k];
	float ak_minus1_local = aGPU[k-1]; 

	for (int i=0; i<n-1; i++) {
		
		//Use of registers
		float ai_local = aGPU[i];
		
		float num = bsqrGPU[i] * (ai_local - ak_minus1_local) * (ai_local - ak_local);
		
		float deno = (ai_local - xGPU_local) * (ai_local - xGPU_local) 
		        * (ai_local - xGPU_local);
		
		sum +=  num / deno;
	}

	float term1 = 3 * xGPU_local - *gammaGPU - ak_local - ak_minus1_local;

	return term1 + sum;
}




// Kernel just for testing sigma interior function
// We can certainly do better in terms of paralellisation
// Also here we calculate also sigma for the two limiting intervals : 
//(a[0], +inf) and (-inf, a[n-1]), a different formula should be used
__global__ void sigma_interior_kernel(float *aGPU, 
				      float *bsqrGPU, 
				      float *yvecGPU, 
				      float *xvecGPU, 
				      float *gammaGPU, 
				      int n) {


	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	while(idx < n){

	yvecGPU[idx] =  sigma_interior(aGPU, bsqrGPU, &(xvecGPU[idx]), gammaGPU, n, idx);
	idx += gridDim.x * blockDim.x;

	}
}



// Interior version for computation of alpha
__device__ float alpha_interior(float *sigma, float *x, float *ak, float *ak_minus1){

	return *sigma / (*ak_minus1 - *x) * (*ak - *x);
}


// Interior version for computation of beta
__device__ float beta_interior(float *fprime, float *f, float *x, float *ak, float *ak_minus1){

	float x_local = *x;
	float fac = (1 / (*ak_minus1 - x_local) + 1 / (*ak - x_local)); 
	return *fprime - fac * (*f); 
	
}





int main (void) {

	// Declare vectors
	float *a, *b, *bsqr, *xvec, *yvec, *c, *gamma;

	// Size of arrow matrix
	int n = 10;
	
	// Memory allocation
	a = (float*)malloc((n-1)*sizeof(float));
	b = (float*)malloc((n-1)*sizeof(float));
	bsqr = (float*)malloc((n-1)*sizeof(float));
	c = (float*)malloc(n*n*sizeof(float));
	xvec = (float*)malloc(n*sizeof(float));
	yvec = (float*)malloc(n*sizeof(float));
	gamma = (float*)malloc(sizeof(float));
	


	// Fill the vectors
	for (int i=0; i<n; i++){
		a[i] = 20 - i;
	}

	for (int i=0; i<n-1; i++){
		b[i] = 10 - i;
	}

	// We take the middle of the intervals
	for (int i=1; i<n-1; i++){
		xvec[i] = (a[i-1] + a[i]) / 2 ;
	}
	
	//Arbitrary filling of the edges values
	xvec[0] = a[0] + 5;
	xvec[n-1] = a[n-2] - 5; 

	//Fill gamma 
	*gamma = 1;


	// Fill c with arrow matrix generated from a and b
	generate_arrow(a, b, c, *gamma, n);

	// Print c
	print_matrix(c, n);

	
	// Declare vectors on GPU
	float *aGPU, *bGPU, *bsqrGPU, *xvecGPU, *yvecGPU, *gammaGPU;

	// Create memory space for vectors on GPU
	hipMalloc(&aGPU, (n-1)*sizeof(float));
	hipMalloc(&bGPU, (n-1)*sizeof(float));
	hipMalloc(&bsqrGPU, (n-1)*sizeof(float));
	hipMalloc(&xvecGPU, n*sizeof(float));
	hipMalloc(&yvecGPU, n*sizeof(float));
	hipMalloc(&gammaGPU, sizeof(float));
	

	// Transfers on GPU
	hipMemcpy(aGPU, a, (n-1)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(bGPU, b, (n-1)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(xvecGPU, xvec, n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(gammaGPU, gamma, sizeof(float), hipMemcpyHostToDevice);


	//Compute square of b on GPU
	square_kernel <<<1024, 512>>> (bGPU, bsqrGPU, n);

	// Transfer on CPU and print to check result
	//cudaMemcpy(bsqr, bsqrGPU, (n-1)*sizeof(float), cudaMemcpyDeviceToHost);
	//print_vector(bsqr, n-1);

	//Compute spectral function on GPU
	//spectral_func_kernel <<<1024, 512>>> (aGPU, bsqrGPU, yvecGPU, xvecGPU, gammaGPU, n);

	// Transfer spectral function results on CPU to print it
	//cudaMemcpy(yvec, yvecGPU, n*sizeof(float), cudaMemcpyDeviceToHost);
	//print_vector(yvec, n);

	//Compute sigma_interior function on GPU
	sigma_interior_kernel <<<1024, 512>>> (aGPU, bsqrGPU, yvecGPU, xvecGPU, gammaGPU, n);

	// Transfer spectral function results on CPU to print it
	hipMemcpy(yvec, yvecGPU, n*sizeof(float), hipMemcpyDeviceToHost);
	print_vector(yvec, n);


	// Free memory on GPU
	hipFree(aGPU);
	hipFree(bGPU);
	hipFree(bsqrGPU);


	// Free memory on CPU
	free(a);
	free(b);
	free(bsqr);
	free(c);
	
}


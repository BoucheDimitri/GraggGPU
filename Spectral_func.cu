
#include <hip/hip_runtime.h>
#include <stdio.h>


//Function to print a small square matrix of floats
void print_matrix(float *c, int n) {

	for (int i=0; i<n; i++){

    		for(int j=0; j<n; j++) {

         		printf("%f     ", c[n * i + j]);
        	}

    		printf("\n");
 	}	
}

//Function to print a small vector of floats
void print_vector(float *c, int n) {

	for (int i=0; i<n; i++){

		printf("%f     ", c[i]);

    		printf("\n");
 	}	
}



// Fill c with arrow matrix generated from vectors a and b
void generate_arrow(float *a, float *b, float *c, float gamma, int n) {
	
	int j = 0; 

	// Fill the arrow
	for (int i=0; i<n; i ++){
		
		if (i<n-1) {

		// Iterate over the last column of c
		c[n - 1 + i*n] = b[i];
		
		// Iterate over the last row of c
		c[n * (n-1) + i] = b[i];

		// Iterate over the diagonal of c
		c[n*i + j] = a[i];
		j ++; 

		}
	}

	// Fill last element of diagonal with gamma
	c[(n-1) * (n-1)] = gamma;
}



// Kernel for computing the square of a vector
// The square of b is needed during several computations 
// for all subproblems, so better to compute it once and for all at the beginning
__global__ void square_kernel(float *bGPU, float *bsqrGPU, int n){
	
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	
	while(idx < n){
	bsqrGPU[idx] = bGPU[idx] * bGPU[idx];
	idx += gridDim.x * blockDim.x;
	}
}



// Device function for computing f at a given point
__device__ float spectral_func(float *aGPU, 
			       float *bsqrGPU, 
			       float x, 
			       float *gammaGPU, 
			       int n) {
	
	float sum = 0;

	for (int i=0; i<n-1; i++){
		sum += bsqrGPU[i] / (aGPU[i] - x);
	}
	
	return x - *gammaGPU + sum;
}


// Device function for computing f' at a given point
__device__ float spectral_func_prime(float *aGPU, 
			       	     float *bsqrGPU, 
			             float x, 
			             int n) {
	
	float sum = 0;

	for (int i=0; i<n-1; i++){

		int ai_local = aGPU[i];
		sum += bsqrGPU[i] / ((ai_local - x) * (ai_local - x));
	}
	
	return 1 + sum;
}





// Kernel associated with spectral_func device function
//__global__ void spectral_func_kernel(float *aGPU, 
//				     float *bsqrGPU, 
//				     float *yvecGPU, 
//				     float *xvecGPU, 
//				     float *gammaGPU, 
//				     int n) {
//
//	int idx = threadIdx.x + blockIdx.x * blockDim.x;
//
//	while(idx < n){
//
//	yvecGPU[idx] = spectral_func(aGPU, bsqrGPU, &(xvecGPU[idx]), gammaGPU, n);
//	idx += gridDim.x * blockDim.x;
//
//	}
//}



// Device function to compute the interior versions of sigma
__device__ float interior_sigma(float *aGPU, 
		       		float *bsqrGPU, 
		                float x, 
		                float *gammaGPU, 
				int k,
		                int n) {
	
	float sum = 0;

	//Use the registers
	float ak_local = aGPU[k];
	float ak_minus1_local = aGPU[k-1]; 

	for (int i=0; i<n-1; i++) {
		
		//Use of registers
		float ai_local = aGPU[i];
		
		float num = bsqrGPU[i] * (ai_local - ak_minus1_local) * (ai_local - ak_local);
		
		float deno = (ai_local - x) * (ai_local - x) 
		        * (ai_local - x);
		
		sum +=  num / deno;
	}

	float term1 = 3 * x - *gammaGPU - ak_local - ak_minus1_local;

	return term1 + sum;
}




// Kernel just for testing sigma interior function
// We can certainly do better in terms of paralellisation
// Also here we calculate also sigma for the two limiting intervals : 
//(a[0], +inf) and (-inf, a[n-1]), a different formula should be used
//__global__ void sigma_interior_kernel(float *aGPU, 
//				      float *bsqrGPU, 
//				      float *yvecGPU, 
//				      float *xvecGPU, 
//				      float *gammaGPU, 
//				      int n) {
//
//
//	int idx = threadIdx.x + blockIdx.x * blockDim.x;
//
//	while(idx < n){

//	yvecGPU[idx] =  interior_sigma(aGPU, bsqrGPU, &(xvecGPU[idx]), gammaGPU, n, idx);
//	idx += gridDim.x * blockDim.x;
//
//	}
//}



// Interior version for computation of alpha
//__device__ float interior_alpha(float *sigma, float *x, float *ak, float *ak_minus1){

//	return *sigma / (*ak_minus1 - *x) * (*ak - *x);
//}


// Interior version for computation of alpha
__device__ float interior_alpha(float sigma, float x, float ak, float ak_minus1){

	return sigma / ((ak_minus1 - x) * (ak - x));
}



// Interior version for computation of beta
__device__ float interior_beta(float fprime, float f, float x, float ak, float ak_minus1){

	float fac = (1 / (ak_minus1 - x) + 1 / (ak - x)); 
	return fprime - fac * f; 
	
}


// Square root device function
__device__ float square_root(float x){

	return expf(0.5 * logf(x));
}


// Computation of the update (delta)
__device__ float interior_delta(float *f, float *alpha, float *beta){


	float term1 = 2 * (*f) / (*beta);
	float term2 = 2 * (*alpha) / (*beta);
	float deno = 1 + square_root(1 + term1 * term2);
	return term1 / deno; 
}



__global__ void test_all_kernel(float *aGPU, 
				float *bsqrGPU, 
				float *yvecGPU, 
				float *xvecGPU, 
				float *gammaGPU, 
				int n) {


	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	// IMPORTANT : n-2 and not n to consider only the interior intervals 
	// TO BE MODIFIED 
	while(idx < n-2){
		float x_local = xvecGPU[idx + 1]; 
		float sig = interior_sigma(aGPU, bsqrGPU, x_local, gammaGPU, idx + 1, n);
		//printf("%f \n", sig); 
		//float alpha = interior_alpha(sig, &(xvecGPU[idx + 1]), aGPU, idx + 1);
		float ak_local = aGPU[idx + 1]; 
		float ak_minus1_local = aGPU[idx]; 
		float alpha = interior_alpha(sig, x_local, ak_local, ak_minus1_local);
		//yvecGPU[idx + 1] = alpha;
		//yvecGPU[idx + 1] = sig;
		float f = spectral_func(aGPU, bsqrGPU, x_local, gammaGPU, n);
		yvecGPU[idx + 1] = alpha;
		idx += gridDim.x * blockDim.x;
	}
}


int main (void) {

	// Declare vectors
	float *a, *b, *bsqr, *xvec, *yvec, *c, *gamma;

	// Size of arrow matrix
	int n = 10;
	
	// Memory allocation
	a = (float*)malloc((n-1)*sizeof(float));
	b = (float*)malloc((n-1)*sizeof(float));
	bsqr = (float*)malloc((n-1)*sizeof(float));
	c = (float*)malloc(n*n*sizeof(float));
	xvec = (float*)malloc(n*sizeof(float));
	yvec = (float*)malloc(n*sizeof(float));
	gamma = (float*)malloc(sizeof(float));
	


	// Fill the vectors
	for (int i=0; i<n; i++){
		a[i] = 20 - i;
	}

	for (int i=0; i<n-1; i++){
		b[i] = 10 - i;
	}

	//Set gamma
	*gamma = 1;

	// We take the middle of the intervals (initial values from the paper for interior points)
	for (int i=1; i<n-1; i++){
		xvec[i] = (a[i-1] + a[i]) / 2 ;
	}
	
	//Arbitrary filling of the edges values (TO REPLACE BY INITIAL VALUES FROM THE PAPER)
	xvec[0] = a[0] + 5;
	xvec[n-1] = a[n-2] - 5; 


	// Fill c with arrow matrix generated from a and b
	generate_arrow(a, b, c, *gamma, n);

	// Print c
	print_matrix(c, n);

	
	// Declare vectors on GPU
	float *aGPU, *bGPU, *bsqrGPU, *xvecGPU, *yvecGPU, *gammaGPU;

	// Create memory space for vectors on GPU
	hipMalloc(&aGPU, (n-1)*sizeof(float));
	hipMalloc(&bGPU, (n-1)*sizeof(float));
	hipMalloc(&bsqrGPU, (n-1)*sizeof(float));
	hipMalloc(&xvecGPU, n*sizeof(float));
	hipMalloc(&yvecGPU, n*sizeof(float));
	hipMalloc(&gammaGPU, sizeof(float));
	

	// Transfers on GPU
	hipMemcpy(aGPU, a, (n-1)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(bGPU, b, (n-1)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(xvecGPU, xvec, n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(gammaGPU, gamma, sizeof(float), hipMemcpyHostToDevice);


	//Compute square of b on GPU
	square_kernel <<<1024, 512>>> (bGPU, bsqrGPU, n);

	// Transfer on CPU and print to check result
	//cudaMemcpy(bsqr, bsqrGPU, (n-1)*sizeof(float), cudaMemcpyDeviceToHost);
	//print_vector(bsqr, n-1);

	//Compute spectral function on GPU
	//spectral_func_kernel <<<1024, 512>>> (aGPU, bsqrGPU, yvecGPU, xvecGPU, gammaGPU, n);

	// Transfer spectral function results on CPU to print it
	//cudaMemcpy(yvec, yvecGPU, n*sizeof(float), cudaMemcpyDeviceToHost);
	//print_vector(yvec, n);

	//Compute sigma_interior function on GPU
	//sigma_interior_kernel <<<1024, 512>>> (aGPU, bsqrGPU, yvecGPU, xvecGPU, gammaGPU, n);

	test_all_kernel <<<1024, 512>>> (aGPU, bsqrGPU, yvecGPU, xvecGPU, gammaGPU, n);

	// Transfer spectral function results on CPU to print it
	hipMemcpy(yvec, yvecGPU, n*sizeof(float), hipMemcpyDeviceToHost);
	print_vector(yvec, n);


	// Free memory on GPU
	hipFree(aGPU);
	hipFree(bGPU);
	hipFree(bsqrGPU);


	// Free memory on CPU
	free(a);
	free(b);
	free(bsqr);
	free(c);
	
}

